
#include <hip/hip_runtime.h>
#ifdef USE_HDF5
/*
 TODO:
 - only load parts of the file, in accordance with a prototxt param "max_mem"
 */

#include <stdint.h>
#include <vector>

#include "hdf5.h"
#include "hdf5_hl.h"

#include "caffe/layers/hdf5_data_layer.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void HDF5DataLayer<Dtype, MItype, MOtype>::Forward_gpu(
                                         const vector<Blob<MItype>*>& bottom,
                                         const vector<Blob<MOtype>*>& top) {
  const int_tp batch_size = this->layer_param_.hdf5_data_param().batch_size();
  for (int_tp i = 0; i < batch_size; ++i) {
    while (Skip()) {
      Next();
    }
    for (int j = 0; j < this->layer_param_.top_size(); ++j) {
      int data_dim = top[j]->count() / top[j]->shape(0);
      vptr<Dtype> top_data = top[j]->mutable_gpu_data() + i * data_dim;
      this->device_->template copy<Dtype>(data_dim,
         &hdf_blobs_[j]->cpu_data()[data_permutation_[current_row_] * data_dim],
         top_data);
    }
    Next();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(HDF5DataLayer);

}  // namespace caffe
#endif  // USE_HDF5
